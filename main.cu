#include "hip/hip_runtime.h"
#include <iostream>

#define STB_IMAGE_IMPLEMENTATION
#include "lib/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image_write.h"

#define CHANNELS 3
#define BLOCK_SIZE 32

const char* inputPath = "../res/input.png";
const char* outputPath = "../res/output.png";

float* loadImageData (const unsigned char* img, unsigned int size);
unsigned char* saveImageData (const float* img, unsigned int size);
__global__ void robertsFilter(const float* input, float* output, int width, int height, int mode);

int main() {
    int width;
    int height;
    int rgb;
    int mode;

    std::cout << "1. Default\n2. Only gx\n3. Only gy\n4. Light mode\nSELECT: ";
    std::cin >> mode;

    std::cout << "\nLoading image...\n";
    unsigned char* image = stbi_load(inputPath, &width, &height, &rgb, CHANNELS);
    float* hostInput = loadImageData(image, width * height * CHANNELS);
    float* hostOutput = (float*)malloc(width * height * sizeof(float));
    float* deviceInput;
    float* deviceOutput;

    unsigned int imageSize = width * height * sizeof(unsigned int);

    hipMalloc((void**)&deviceInput, imageSize);
    hipMalloc((void**)&deviceOutput, imageSize);
    hipMemcpy(deviceInput, hostInput, imageSize, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    std::cout << "Processing image...\n";
    robertsFilter<<<gridSize, blockSize>>>(deviceInput, deviceOutput, width, height, mode);
    hipMemcpy(hostOutput, deviceOutput, imageSize, hipMemcpyDeviceToHost);

    std::cout << "Saving result...\n";
    stbi_write_png(outputPath, width, height, CHANNELS,
                   saveImageData(hostOutput, width * height), width * CHANNELS);

    stbi_image_free(image);

    hipFree(deviceInput);
    hipFree(deviceOutput);

    free(hostInput);
    free(hostOutput);

    std::cout << "Done!\n";
    return 0;
}

__global__ void robertsFilter(const float* input, float* output, int width, int height, int mode) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width - 1 && y < height - 1) {
        float gx = input[y * width + x] - input[(y+1) * width + (x+1)];
        float gy = input[y * width + (x+1)] - input[(y+1) * width + x];

        if(mode == 2)
            gy = 0.0;
        if(mode == 3)
            gx = 0.0;

        double magnitude = sqrtf((float)(gx * gx + gy * gy));

        // Light mode
        if(mode == 4)
        {
            magnitude = 1 - magnitude;
        }

        output[y * width + x] = (float)(magnitude);
    }

}

float* loadImageData (const unsigned char* img, unsigned int size) {
    float* data = new float[size / CHANNELS];

    int sum = 0;

    for (unsigned int i = 0; i < size; ++i) {
        sum += img[i];
        if ((i + 1) % CHANNELS == 0) {
            data[i / CHANNELS] = (sum / 3.f) / 255.f;
            sum = 0;
        }
    }

    return data;
}

unsigned char* saveImageData (const float* img, unsigned int size) {
    unsigned char* data = new unsigned char[size * CHANNELS];

    for (unsigned int i = 0; i < size; ++i) {
        for (unsigned int j = 0; j < CHANNELS; ++j) {
            data[i * CHANNELS + j] = img[i] * 255;
        }
    }

    return data;
}
