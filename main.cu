#include "hip/hip_runtime.h"
#include <iostream>

#define STB_IMAGE_IMPLEMENTATION
#include "lib/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image_write.h"

#define CHANNELS 3
#define BLOCK_SIZE 32

const char* inputPath = "../res/input2.png";
const char* outputPath = "../res/output.png";

unsigned int* loadImageData (const unsigned char* img, unsigned int size);
unsigned char* saveImageData (const unsigned int* img, unsigned int size);
__global__ void robertsFilter(const unsigned int* input, unsigned int* output, int width, int height);

int main() {
    int width;
    int height;
    int rgb;

    unsigned char* image = stbi_load(inputPath, &width, &height, &rgb, CHANNELS);
    unsigned int* hostInput = loadImageData(image, width * height * CHANNELS);
    unsigned int* hostOutput = (unsigned int*)malloc(width * height * sizeof(unsigned int));
    unsigned int* deviceInput;
    unsigned int* deviceOutput;

    unsigned int imageSize = width * height * sizeof(unsigned int);

    hipMalloc((void**)&deviceInput, imageSize);
    hipMalloc((void**)&deviceOutput, imageSize);
    hipMemcpy(deviceInput, hostInput, imageSize, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    robertsFilter<<<gridSize, blockSize>>>(deviceInput, deviceOutput, width, height);
    //hipMemcpy(deviceInput, deviceOutput, imageSize, hipMemcpyDeviceToDevice);

    hipMemcpy(hostOutput, deviceOutput, imageSize, hipMemcpyDeviceToHost);

    stbi_write_png(outputPath, width, height, CHANNELS,
                   saveImageData(hostOutput, width * height), width * CHANNELS);

    stbi_image_free(image);

    hipFree(deviceInput);
    hipFree(deviceOutput);

    free(hostInput);
    free(hostOutput);

    return 0;
}

__global__ void robertsFilter(const unsigned int* input, unsigned int* output, int width, int height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width - 1 && y < height - 1) {
        unsigned int gx = input[(y+1) * width + x+1] - input[y * width + x];
        unsigned int gy = input[(y+1) * width + x] - input[y * width + x+1];
        double magnitude = sqrtf((float)(gx * gx + gy * gy));

        output[y * width + x] = (unsigned char)(magnitude);
    }
}

unsigned int* loadImageData (const unsigned char* img, unsigned int size) {
    unsigned int* data = new unsigned int[size / CHANNELS];

    int sum = 0;

    for (unsigned int i = 0; i < size; ++i) {
        sum += img[i];
        if ((i + 1) % CHANNELS == 0) {
            data[i / CHANNELS] = sum > 0 ? 1 : 0;
            sum = 0;
        }
    }

    return data;
}

unsigned char* saveImageData (const unsigned int* img, unsigned int size) {
    unsigned char* data = new unsigned char[size * CHANNELS];

    for (unsigned int i = 0; i < size; ++i) {
        for (unsigned int j = 0; j < CHANNELS; ++j) {
            data[i * CHANNELS + j] = img[i] * 255;
        }
    }

    return data;
}
